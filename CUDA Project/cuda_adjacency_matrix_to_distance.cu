
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

#define N 128
#define BLOCK_SIZE 8
#define THREAD_COUNT 32
#define FILE_NAME "output.txt"

#define WRITE_MATRIX(M, X, Y, V) \
  M[X+(Y*N)] = V;

#define READ_MATRIX(M, X, Y, R) \
  R = M[X+(Y*N)];

void printMatrix(int*, ofstream&);
bool checkCircularResults(int*);
bool checkLinearResults(int*);
__global__ void calculateDistanceMatrix(int*, int);
__global__ void initializeCircularGraph(int*);
__global__ void initializeLinearGraph(int*);

int main(){
  // first, make sure N is even and in range
  if (N%2 != 0){
    cout << "ERROR: N must be even. Change N and recompile!" << endl;
    return 1;
  } else if (N > 128){
    cout << "ERROR: The max size of N is 128. Change N and recompile!" << endl;
    return 1;
  }

  // variables
  int *matrix;
  int *d_matrix;
  int k;
  int size = N * N * sizeof(int);
  ofstream myFile;

  myFile.open(FILE_NAME);

  // Allocate space for device copy of matrix
  hipMalloc((void **)&d_matrix, size);

  // Allocate space for host copy of matrix
  matrix = (int *)malloc(size);

  // Copy matrix to device and initialize
  hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);
  initializeCircularGraph<<<BLOCK_SIZE,THREAD_COUNT>>>(d_matrix);

  // Copy initialized matrix to host
  hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

  // do cuda stuff
  // NOTE: CUDA does not have any way to synchronize blocks. Because we need
  // all blocks to by synced after each K step, the way to accomplish this is
  // to end the kernel whenever block synchronization is required.
  for (k=0; k<N; k++){
    calculateDistanceMatrix<<<BLOCK_SIZE,THREAD_COUNT>>>(d_matrix, k);
  }

  myFile << "                          ADJACENCY MATRIX" << endl;
  myFile << "==============================================================================" << endl;
  printMatrix(matrix, myFile);

  // Copy results back to host
  hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

  myFile << "                            DISTANCE MATRIX" << endl;
  myFile << "==============================================================================" << endl;
  printMatrix(matrix, myFile);

  myFile << "RESULT OF MATRIX CHECK: " << checkCircularResults(matrix) << endl;

  // Cleanup
  free(matrix);
  hipFree(d_matrix);
  myFile.close();

  return 0;
}

void printMatrix(int *matrix, ofstream &myFile){
  int i;

  for (i=0; i<N*N; i++){
    if (i%N == 0){
      myFile << endl;
    }
    myFile << matrix[i] << "\t";
  }
  myFile << endl;
}

bool checkCircularResults(int *matrix){
  int x, y, number, n;
  bool increment = true;

  number = 0;

  for (y=0; y<N; y++){
    for (x=0; x<N; x++){
      READ_MATRIX(matrix, x, y, n);
      if (number != n){
        cout << "Matrix check failed!" << endl;
        cout << "These matrix coordinates failed:" << endl;
        cout << "X: " << x << " Y: " << y << endl;
        cout << "Found " << n << " where " << number << " was expected" << endl;
        return false;
      }
      if (number == N/2){
        increment = false;
      } else if (number == 0){
        increment = true;
      }
      if (increment == true){
        number ++;
      } else {
        number --;
      }
    }
    if (y < N/2){
      number++;
    } else {
      number--;
    }
  }
  cout << "Matrix check passed!" << endl;
  return true;
}

__global__ void calculateDistanceMatrix(int *matrix, int k){
  int i, j, num1, num2, num3;

  for (j=0; j<N; j++){
    if ((j%BLOCK_SIZE) == blockIdx.x){
      for (i=0; i<N; i++){
        if ((i%THREAD_COUNT) == threadIdx.x){
          READ_MATRIX(matrix, i, j, num1);
          READ_MATRIX(matrix, i, k, num2);
          READ_MATRIX(matrix, k, j, num3);
          int minimum = min(num1, num2 + num3);
          WRITE_MATRIX(matrix, i, j, minimum);
        }
      }
    }
  }



  // divide rows up by block, and columns up by threads

  // since we are restricted to 8 blocks with 32 threads per block,
  // my strategy for a 100x100 matrix would be:

  // block 0 grabs row 0 and thread 0 would be responsible for the
  // 0th, 33rd, 66th, and 99th column, while process 2 would only be
  // responsible for the 2nd, 35th, and 68th column

  // At the same time, block 0 would be responsible for the 0th, 9th,
  // 18th, 27th, 36th, 45th, 54th, 63rd, 72nd, 81st, 90th, and 99th rows

  // As each row is completed, the matrix for that row will be updated, and
  // the block will move on to the next one. Once all blocks are finished,
  // that signals the completion of the kth step, and the blocks will update
  // the rows of the matrix in device memory that they were responsible for
  // and then synchronize blocks
  // By doing this, no race conditions should exist

  // This process will go from k = 0 to k = N-1 steps, and once all these
  // k steps are done, the function will be complete

}

__global__ void initializeCircularGraph(int *matrix){
  int i, j;

  for (j=0; j<N; j++){
    if ((j%BLOCK_SIZE) == blockIdx.x){
      for (i=0; i<N; i++){
        if ((i%THREAD_COUNT) == threadIdx.x){
          // diagonal is zero
          if (i == j){
            WRITE_MATRIX(matrix, i, j, 0);
          // makes graph linear
          } else if (i == j+1 || j == i+1){
            WRITE_MATRIX(matrix, i, j, 1);
          // makes graph circular
          } else if((i == 0 && j == N-1) || (j == 0 && i == N - 1)){
            WRITE_MATRIX(matrix, i, j, 1);
          // every other node isn't connected
          } else {
            WRITE_MATRIX(matrix, i, j, 999999)
          }
        }
      }
    }
  }
}

__global__ void initializeLinearGraph(int *matrix){
  int i, j;

  for (j=0; j<N; j++){
    if ((j%BLOCK_SIZE) == blockIdx.x){
      for (i=0; i<N; i++){
        if ((i%THREAD_COUNT) == threadIdx.x){
          // diagonal is zero
          if (i == j){
            WRITE_MATRIX(matrix, i, j, 0);
          // makes graph linear
          } else if (i == j+1 || j == i+1){
            WRITE_MATRIX(matrix, i, j, 1);
          // every other node isn't connected
          } else {
            WRITE_MATRIX(matrix, i, j, 999999)
          }
        }
      }
    }
  }
}
